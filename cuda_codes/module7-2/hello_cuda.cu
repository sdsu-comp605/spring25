
#include <hip/hip_runtime.h>
#include <stdio.h>

// empty function kernel() qualified with __global__
__global__ void kernel(void)
{
}

int main(void)
{
    // A call to the empty function, with additional <<<1,1>>>
    kernel<<<1,1>>>();

    printf("Hello, World! \n");
    return 0;
}
